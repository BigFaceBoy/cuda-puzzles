#include "hip/hip_runtime.h"
/**
 * Puzzle 6
 * Implement a kernel that adds 10 to each position of a and stores it in out. You have fewer threads per block than the size of a.
 * 
*/

#include <utils.h>
__global__ void vector_add10(int* a, int *out, int size){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // 网格跨步循环处理多个元素
    for(int i = idx; i < size; i += stride) {
        out[i] = a[i] + 10;  // 无分支连续内存访问
    }
}


#define SIZE 1000
int main(){

    int* h_a = (int*)malloc(sizeof(int) * SIZE);
    int* h_b = (int*)malloc(sizeof(int) * SIZE);
    int* h_out = (int*)malloc(sizeof(int) * SIZE);
    for(int i = 0; i < SIZE; i++){
        h_a[i] = i;
        h_b[i] = i+1;
    }
    

    int *d_a;
    int *d_b;
    int *d_out;
    hipMalloc(&d_a, sizeof(int) * SIZE);
    hipMalloc(&d_b, sizeof(int) * SIZE);
    hipMalloc(&d_out, sizeof(int) * SIZE);
    hipMemcpy(d_a, h_a, sizeof(int) * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int) * SIZE, hipMemcpyHostToDevice);

    int THREAD_NUM_PER_BLOCK = 64;
    int block_num = (SIZE + THREAD_NUM_PER_BLOCK - 1) / THREAD_NUM_PER_BLOCK;

    vector_add_vector<<<block_num, THREAD_NUM_PER_BLOCK>>>(d_a,d_b, d_out, SIZE); 

    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, sizeof(int) * SIZE, hipMemcpyDeviceToHost);
    printVector(h_out, SIZE, 10);
}