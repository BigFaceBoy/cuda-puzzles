#include "hip/hip_runtime.h"
/**
 * Puzzle 2
 * Implement a kernel that adds together each position of a and b and stores it in out. You have 1 thread per position
*/

#include <utils.h>
__global__ void vector_add_vector(int* a,int* b, int *out, int size){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
}

#define SIZE 1000
int main(){

    int* h_a = (int*)malloc(sizeof(int) * SIZE);
    int* h_b = (int*)malloc(sizeof(int) * SIZE);
    int* h_out = (int*)malloc(sizeof(int) * SIZE);
    for(int i = 0; i < SIZE; i++){
        h_a[i] = i;
        h_b[i] = i+1;
    }
    

    int *d_a;
    int *d_b;
    int *d_out;
    hipMalloc(&d_a, sizeof(int) * SIZE);
    hipMalloc(&d_b, sizeof(int) * SIZE);
    hipMalloc(&d_out, sizeof(int) * SIZE);
    hipMemcpy(d_a, h_a, sizeof(int) * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int) * SIZE, hipMemcpyHostToDevice);

    int THREAD_NUM_PER_BLOCK = 64;
    int block_num = (SIZE + THREAD_NUM_PER_BLOCK - 1) / THREAD_NUM_PER_BLOCK;

    vector_add_vector<<<block_num, THREAD_NUM_PER_BLOCK>>>(d_a,d_b, d_out, SIZE); 

    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, sizeof(int) * SIZE, hipMemcpyDeviceToHost);
    // 0 1 2 3 4 5 6 7 8 9
    // 1 2 3 4 5 6 7 8 9 10
    printVector(h_out, SIZE, 10);
}