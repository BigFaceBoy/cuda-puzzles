#include "hip/hip_runtime.h"
/**
 * Puzzle 1
 * Implement a kernel that adds 10 to each position of vector a and stores it in vector out. You have 1 thread per position.
*/

#include <utils.h>
__global__ void vector_add10(int* a, int *out, int size){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    out[idx] = a[idx] + 10;
}

#define SIZE 1024
int main(){
    int* h_a = (int*)malloc(sizeof(int) * SIZE);
    int* h_out = (int*)malloc(sizeof(int) * SIZE);
    for(int i = 0; i < SIZE; i++){
        h_a[i] = i;
    }

    int *d_a;
    int *d_out;
    hipMalloc(&d_a, sizeof(int) * SIZE);
    hipMalloc(&d_out, sizeof(int) * SIZE);
    hipMemcpy(d_a, h_a, sizeof(int) * SIZE, hipMemcpyHostToDevice);

    int THREAD_NUM_PER_BLOCK = 64;
    int block_num = (SIZE + THREAD_NUM_PER_BLOCK - 1) / THREAD_NUM_PER_BLOCK;

    vector_add10<<<block_num, THREAD_NUM_PER_BLOCK>>>(d_a, d_out, SIZE); 

    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, sizeof(int) * SIZE, hipMemcpyDeviceToHost);
    printVector(h_out, SIZE, 10);
}

