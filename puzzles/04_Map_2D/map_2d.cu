#include "hip/hip_runtime.h"
/**
 * Puzzle 4
 * Implement a kernel that adds 10 to each position of a and stores it in out. Input a is 2D and square. You have more threads than positions.
 * 
*/

#include <utils.h>
__global__ void matrix_add10(int* a, int *out, int M, int N){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x < M && y < N){
        out[y * M + x] = a[y * M + x] + 10;
    }
}

#define M 1024
#define N 1024
#define SIZE (M * N)
int main(){
    int* h_a = (int*)malloc(sizeof(int) * SIZE);
    int* h_out = (int*)malloc(sizeof(int) * SIZE);
    for(int i = 0; i < SIZE; i++){
        h_a[i] = i;
    }

    int *d_a;
    int *d_out;
    hipMalloc(&d_a, sizeof(int) * SIZE);
    hipMalloc(&d_out, sizeof(int) * SIZE);
    hipMemcpy(d_a, h_a, sizeof(int) * SIZE, hipMemcpyHostToDevice);

    int THREAD_NUM_PER_BLOCK = 64;
    int block_num = (SIZE + THREAD_NUM_PER_BLOCK - 1) / THREAD_NUM_PER_BLOCK;

    matrix_add10<<<block_num, THREAD_NUM_PER_BLOCK>>>(d_a, d_out, M, N); 

    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, sizeof(int) * SIZE, hipMemcpyDeviceToHost);
    printMatrix(h_out, SIZE, 10);
}