#include "hip/hip_runtime.h"
/**
 * Puzzle 3
 * Implement a kernel that adds 10 to each position of a and stores it in out. You have more threads than positions.
 * 
*/

#include <utils.h>
__global__ void vector_add10(int* a, int *out, int size){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if(idx < size){
        out[idx] = a[idx] + 10;
    }
}

#define SIZE 1024
int main(){
    int* h_a = (int*)malloc(sizeof(int) * SIZE);
    int* h_out = (int*)malloc(sizeof(int) * SIZE);
    for(int i = 0; i < SIZE; i++){
        h_a[i] = i;
    }

    int *d_a;
    int *d_out;
    hipMalloc(&d_a, sizeof(int) * SIZE);
    hipMalloc(&d_out, sizeof(int) * SIZE);
    hipMemcpy(d_a, h_a, sizeof(int) * SIZE, hipMemcpyHostToDevice);

    int THREAD_NUM_PER_BLOCK = 64;
    int block_num = (SIZE + THREAD_NUM_PER_BLOCK - 1) / THREAD_NUM_PER_BLOCK;

    vector_add10<<<block_num, THREAD_NUM_PER_BLOCK>>>(d_a, d_out, SIZE); 

    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, sizeof(int) * SIZE, hipMemcpyDeviceToHost);
    printVector(h_out, SIZE, 10);
}